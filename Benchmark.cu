
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>

#define NUM_THREADS_PER_BLK 128
//# define ONE_THREAD_PER_BLOCK

void barf(const char *fmt, ...)
{
	va_list args;

	va_start(args, fmt);
	vfprintf(stderr, fmt, args);
	va_end(args);

	exit(1);
}

unsigned long utime(void)
{
    struct timeval tv;
    unsigned long result = 0;

    gettimeofday(&tv, NULL);
    result += (tv.tv_sec * 1000000);
    result += tv.tv_usec;

    return result;
}


__host__ __device__ int getIndex(int rows, int cols, int row, int col) {
    return row * cols + col;
}


__global__ void cuMatrix(float *matrix_a, float *matrix_b, float *out, int rows_a, int cols_a, int rows_b, int cols_b) {

#ifdef ONE_THREAD_PER_BLOCK
		int i = blockIdx.x;
		int j = blockIdx.y;
#else
    int i = blockIdx.x * NUM_THREADS_PER_BLK + threadIdx.x;
    int j = blockIdx.y;

    if (i >= cols_b) {
        return;
    }
#endif

    //int i = blockIdx.y * blockDim.y + threadIdx.y ;   // Row i of matrix C
    //int j = blockIdx.x ;   // Column j of matrix C

    if(i*j > rows_a*cols_b ){return;}
		
    int out_index = i * cols_b + j;
	int a_index, b_index;
	float accum = 0;


    accum = 0;
    for(int k = 0; k < rows_b; k++){
        a_index = getIndex(rows_a, cols_a, i, k);
        b_index = getIndex(rows_b, cols_b, k, j);
        accum+= matrix_a[a_index] * matrix_b[b_index];
    }
	
	  out[out_index] = accum;
}


void allocMatrix(float *ptr, int rows, int cols) {
    for(int i = 0; i < cols*rows; i++) {
        ptr[i] = (float)drand48();
    }
}

void printMatrix(float *matrix, int rows, int cols) {
    printf("[");
    for(int i=0; i < rows; i++) {
        for(int j = 0; j < cols; j++){
            if(j == (cols-1) && i == (rows-1)) {
                printf("%0.4f ]\n", matrix[getIndex(rows, cols, i, j)]);
                return;
            }
            printf("%0.4f ", matrix[getIndex(rows, cols, i, j)]);

        }
        printf("\n ");
    }
}

long gpu_bmark(int N, int M)
{
    srand48(time(0));

    int rows_a = N;
    int cols_a = M;

    int rows_b = cols_a; //rows of b must equal cols of a
    int cols_b = rows_a;

    //create and allocate CPU matrices.
    float *matrix_a = (float*)malloc(rows_a*cols_a*sizeof(float));
    float *matrix_b = (float*)malloc(rows_b*cols_b*sizeof(float));
    float *matrix_res = (float*)malloc(rows_a*cols_b*sizeof(float));

    allocMatrix(matrix_a, rows_a, cols_a);
    allocMatrix(matrix_b, rows_b, cols_b);

    //printMatrix(matrix_a, rows_a, cols_a);
    //printf("\n");
    //printMatrix(matrix_b, rows_b, cols_b);
    //printf("\n");

	for(int i=0; i<(rows_a*cols_b); i++) {
	   matrix_res[i] = 0;
	}

	float *dev_matrix_a, *dev_matrix_b, *dev_matrix_out;
	hipMalloc( (void **) &dev_matrix_a, sizeof(float)*(rows_a)*(cols_a) );
	hipMalloc( (void **) &dev_matrix_b, sizeof(float)*(rows_b)*(cols_b) );
	hipMalloc( (void **) &dev_matrix_out, sizeof(float)*(rows_a)*(cols_b));

	// copy matrix data to device
	hipMemcpy( dev_matrix_a, matrix_a, sizeof(float)*(rows_a)*(cols_a), hipMemcpyHostToDevice );
	hipMemcpy( dev_matrix_b, matrix_b, sizeof(float)*(rows_b)*(cols_b), hipMemcpyHostToDevice );



    // compute grid dimensions
#ifdef ONE_THREAD_PER_BLOCK
    int num_blocks_x = cols_b;
    int num_threads = 1;
#else
	int num_blocks_x = cols_b / NUM_THREADS_PER_BLK;

	if ( cols_b % NUM_THREADS_PER_BLK != 0) {
		num_blocks_x++;
	}
    int num_threads = NUM_THREADS_PER_BLK;
#endif
	//dim3 grid(num_blocks_x, 1);

    dim3 grid(num_blocks_x, rows_a);

	unsigned long begin = utime();

	cuMatrix<<<grid, num_threads>>>( dev_matrix_a, dev_matrix_b, dev_matrix_out, rows_a, cols_a, rows_b, cols_b);

    // copy transformed matrix data from device
	hipMemcpy( matrix_res, dev_matrix_out, sizeof(float)*(rows_a)*(cols_b), hipMemcpyDeviceToHost );

   //printMatrix(matrix_res, rows_a, cols_b);

   hipFree(dev_matrix_a);
   hipFree(dev_matrix_b);
   hipFree(dev_matrix_out);

   unsigned long end = utime();

   unsigned long elapsed = end - begin;
  // printf("computation took %lu microseconds\n", elapsed);

   free(matrix_a);
   free(matrix_b);
   free(matrix_res);
	return elapsed;
}


long cpu_bmark(int N, int M)
{

    int rows_a = N;
    int cols_a = M;

    int rows_b = cols_a;
    int cols_b = rows_a;

    float *matrix_a = (float*)malloc(rows_a*cols_a*sizeof(float));
    float *matrix_b = (float*)malloc(rows_b*cols_b*sizeof(float));
    float *matrix_res = (float*)malloc(rows_a*cols_b*sizeof(float));

    allocMatrix(matrix_a, rows_a, cols_a);
    allocMatrix(matrix_b, rows_b, cols_b);

    //printMatrix(matrix_a, rows_a, cols_a);
    // printf("\n");
    // printMatrix(matrix_b, rows_b, cols_b);
    // printf("\n");


    unsigned long begin = utime();

    int res_index, a_index, b_index;
    for(int i=0; i < rows_a; i++){
        for(int j=0; j < cols_b; j++){
            res_index = getIndex(rows_a, cols_b, i, j);
            matrix_res[res_index] = 0;
            for(int k=0; k < rows_b; k++){
                a_index = getIndex(rows_a,cols_a, i, k);
                b_index = getIndex(rows_b, cols_b, k, j);
                matrix_res[res_index] += matrix_a[a_index] * matrix_b[b_index];
            }
        }
    }
    //printMatrix(matrix_res, rows_a, cols_b);


    unsigned long end = utime();

    unsigned long elapsed = end - begin;

    free(matrix_a);
    free(matrix_b);
    free(matrix_res);
    return elapsed;
}

int main(int argc, char **argv) {
    printf("input size, GPU time\n");
    for(int i=5000; i<=8000; i+=250) {
        printf("%i, %li\n",i, gpu_bmark(i,i));
    }
}
